#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

#define SIZE 10

__global__ void max(int *c,int *a)
{
	int i=threadIdx.x;
	*a=c[0];
	if(c[i]>*a)
	{
		*a=c[i];
	}
}

__global__ void min(int *c,int *a)
{
	int i=threadIdx.x;
	*a=c[0];
	if(c[i]<*a)
	{
		*a=c[i];
	}
}

int main(void)
{
	int i;
	srand(time(NULL));
	int a[SIZE],max_val,min_val;

	int *dev_a,*dev_max,*dev_min;

	hipMalloc((void**)&dev_a, SIZE*sizeof(int));

	hipMalloc((void**)&dev_max,SIZE*sizeof(int));

	hipMalloc((void**)&dev_min,SIZE*sizeof(int));



	for(i=0;i<SIZE;i++)
	{

		a[i] = rand()%20+1;
	}

	printf("\nVector is:\n");
	for(i=0;i<SIZE;i++)
	{
		printf("%d  ",a[i]);
	}

	hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
	max<<<1,SIZE>>>(dev_a,dev_max);
	hipMemcpy(&max_val,dev_max,sizeof(max_val),hipMemcpyDeviceToHost);

	printf("\n\n");
    printf("The maximum value of result vector is:- %d",max_val);

    hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
    min<<<1,SIZE>>>(dev_a,dev_min);
    hipMemcpy(&min_val,dev_min,sizeof(min_val),hipMemcpyDeviceToHost);

    	printf("\n\n");
        printf("The minimum value of result vector is:- %d",min_val);

	return 0;
}
/*

Vector is:
7  6  7  2  18  8  17  13  16  19

The maximum value of result vector is:- 19

The minimum value of result vector is:- 2

 */
