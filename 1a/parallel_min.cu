#include "hip/hip_runtime.h"
%%cu
#include<hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#define SIZE 16
using namespace std;

__global__ void sum(int* input)
{
	const int tid = threadIdx.x;

	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{
			const int fst = tid * step_size * 2;
			const int snd = fst + step_size;
			if(input[fst]>input[snd])
			input[fst]=input[snd];
		}

		step_size <<= 1;
		number_of_threads >>= 1;
	}
}

int main()
{
	int count = SIZE;
	const int size = count*sizeof(int);
	int h[SIZE];
	for(int i=0;i<count;i++)
	{
	h[i]=rand()%50;
	}
	for(int i=0;i<count;i++)
	{
	cout<<" "<<h[i];
	}

	int* d;

	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	sum <<<1, count/2>>>(d);

	int result;
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);

	cout << "Min is " << result << endl;

	//getchar();

	hipFree(d);
	

	return 0;
}

