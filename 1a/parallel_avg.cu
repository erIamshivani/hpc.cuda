#include "hip/hip_runtime.h"
%%cu
#include<hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#define SIZE 15
using namespace std;

__global__ void sum(int* input)
{
	const int tid = threadIdx.x;

	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{
			const int fst = tid * step_size * 2;
			const int snd = fst + step_size;
			input[fst]+=input[snd];
		}

		step_size <<= 1;
		number_of_threads >>= 1;
	}
	input[0]=input[0]/SIZE;
}

int main()
{
	int count = SIZE;
	const int size = count*sizeof(int);
	int h[SIZE];
	for(int i=0;i<count;i++)
	{
	h[i]=rand()%50;
	}
	for(int i=0;i<count;i++)
	{
	cout<<" "<<h[i];
	}

	int* d;

	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	sum <<<1, count>>>(d);

	int result;
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);

	cout << "AVG is " << result << endl;

	//getchar();

	hipFree(d);
	

	return 0;
}

