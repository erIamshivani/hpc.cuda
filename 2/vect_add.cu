#include <stdio.h>
#include<hip/hip_runtime.h>
#include <stdlib.h>
#include<time.h>

#define SIZE 20

__global__ void vectsum(int *x,int *y,int *z)
{
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	z[tid]=x[tid]+y[tid];
}

int main(void)
{
	int i;
	// srand(time(NULL));
	int a[SIZE],b[SIZE],c[SIZE];

	int *dev_a,*dev_b,*dev_c;

	hipMalloc((void **)&dev_a, SIZE*sizeof(int));
	hipMalloc((void **)&dev_b, SIZE*sizeof(int));
	hipMalloc((void **)&dev_c, SIZE*sizeof(int));

	for(i=0;i<SIZE;i++)
	{
		a[i] = rand()%20+1;
	}

	printf("\nThe 1st vector is:\n");
	for(i=0;i<SIZE;i++)
	{
		printf("%d  ",a[i]);
	}

	for(i=0;i<SIZE;i++)
	{
		b[i] = rand()%20+1;
	}

	printf("\nThe 2nd vector is:\n");
	for(i=0;i<SIZE;i++)
	{
		printf("%d  ",b[i]);
	}

	hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,sizeof(b),hipMemcpyHostToDevice);
	vectsum<<<1,SIZE>>>(dev_a,dev_b,dev_c);
	hipMemcpy(&c,dev_c,sizeof(c),hipMemcpyDeviceToHost);

	printf("\nThe result is:\n");
	for(int i=0;i<SIZE;i++)
	{
		printf("%d  ",c[i]);
	}


	return 0;
}
