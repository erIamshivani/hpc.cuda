#include<hip/hip_runtime.h>
#include<stdio.h>

int main(void) {
    void MatrixMultiplication(int *,int *, int *, int);
    const int Width = 3;
    int M[Width*Width], N[Width*Width], P[Width*Width];
    for(int i = 0; i < (Width*Width) ; i++) {
        M[i] = rand()%20 +3;
        N[i] = rand()%20 +4;
        P[i] = 0;
    }
    printf("First Matrix:\n");
    for(int i = 0; i < (Width*Width) ; i=i+Width) {
    for(int j = 0; j < (Width) ; j++) {
            printf("%d ", M[i+j]);
        }
   
    }
    printf("\n");
    printf("Second Matrix:\n");
    for(int i = 0; i < (Width*Width) ; i=i+Width) {
        for(int j = 0; j < (Width) ; j++) {
                printf("%d ", N[i+j]);
            }
       
        }
    printf("\n");
    printf("Multiplication :\n");
    MatrixMultiplication(M, N, P, Width);
    for(int i = 0; i < (Width*Width) ; i=i+Width) {
        for(int j = 0; j < (Width) ; j++) {
                printf("%d ", P[i+j]);
            }
     
        }
    int quit;
    scanf("%d",&quit);
    return 0;
}

//Matrix multiplication kernel - thread specification
__global__ void MatrixMulKernel(int *Md, int *Nd, int *Pd, int Width) {
    //2D Thread ID
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    //Pvalue stores the Pd element that is computed by the thread
    int Pvalue = 0;

    for(int k = 0; k < Width ; ++k) 
    {
        int Mdelement = Md[ty*Width + k];
        int Ndelement = Nd[k*Width + tx];
        Pvalue += (Mdelement*Ndelement);
    }

    Pd[ty*Width + tx] = Pvalue;
}

void MatrixMultiplication(int *M, int *N, int *P, int Width) 
{
    int size = Width*Width*sizeof(int);
    int *Md, *Nd, *Pd;

    //Transfer M and N to device memory
    hipMalloc((void**)&Md, size);
    hipMemcpy(Md,M,size,hipMemcpyHostToDevice);
    hipMalloc((void**)&Nd, size);
    hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);

    //Allocate P on the device
    hipMalloc((void**)&Pd,size);

    //Setup the execution configuration
    dim3 dimBlock(Width,Width);
    dim3 dimGrid(1,1);

    //Launch the device computation threads!
    MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,Width);

    //Transfer P from device to host
    hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);

    //Free device matrices
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
}
